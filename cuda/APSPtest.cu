#include <iostream>
#include <cstdlib>
#include <sys/time.h>

#include <hip/hip_runtime.h>

#include "MatUtil.h"
#include "debug.h"

#if (APSP_VER == 1)
    #include "Floyd.h"
#elif (APSP_VER == 2)
    #include "Floyd_coa.h"
#elif (APSP_VER == 3)
    #include "Floyd_row.h"
#else
    #include "Floyd_blk.h"
#endif

using namespace std;

int main(int argc, char **argv)
{
    struct timeval tv1, tv2;

    if (argc != 2)
    {
        cout << "Usage: test {N}" << endl;
        exit(-1);
    }

    //generate a random matrix.
    size_t N = atoi(argv[1]);
    /*if (N % TILE_WIDTH != 0)*/
    /*{*/
        /*cout << "The problem size must be divisible by " << TILE_WIDTH << endl;*/
        /*exit(-1);*/
    /*}*/

    int *mat = (int*)malloc(sizeof(int)*N*N);
    GenMatrix(mat, N);

    //compute the reference result.
    int *ref = (int*)malloc(sizeof(int)*N*N);
    memcpy(ref, mat, sizeof(int)*N*N);
    gettimeofday(&tv1, NULL);
    ST_APSP(ref, N);
    gettimeofday(&tv2, NULL);
    long sequentialtime = (tv2.tv_sec - tv1.tv_sec)*1000000 + tv2.tv_usec - tv1.tv_usec;
    cout << "Elapsed time (sequential) = " << sequentialtime << " usecs" << endl;

    //compute your results
    int *result = (int*)malloc(sizeof(int)*N*N);
    memcpy(result, mat, sizeof(int)*N*N);
    //replace by parallel algorithm
    gettimeofday(&tv1, NULL);
    Floyd_Warshall(result, N);
    gettimeofday(&tv2, NULL);
    long paralleltime = (tv2.tv_sec - tv1.tv_sec)*1000000 + tv2.tv_usec - tv1.tv_usec;
    cout << "Elapsed time (parallel) = " << paralleltime << " usecs" << endl;

    cout << "Speed up = " << (double)sequentialtime/paralleltime << endl;
    //compare your result with reference result
    if(CmpArray(result, ref, N*N))
        cout << "Your result is correct." << endl;
    else
        cout << "Your result is wrong." << endl;
}
